/*//////////////////////////////////////////////////////////////////////
//                                       _   __                       //
//                             _   __   / | / /                       //
//                   ___      / | / /  /  |/ /                        //
//                  /   |    /  |/ /  / /|  /                         //
//                 / /| |   / /|  /  /_/ |_/ E T W O R K              //
//                / ___ |  /_/ |_/ E  U  R  A  L                      //
//               /_/  |_| R T I F I C I A L                           //
//                                                                    //
//                        ( computation.cu )                          //
//                                                                    //
////////////////////////////////////////////////////////////////////////
///-------------- Copyright © 2016 Krzysztof Baranski  --------------///
//////////////////////////////////////////////////////////////////////*/
#include "computation.h"

__host__ void compute_first() {
	// struct timeval start, stop, diff;
	// gettimeofday(&start, NULL);

	double* d_INPUT;
	double* d_FIRST_LAYER_WEIGHT;
	double* d_FIRST_LAYER;
	
	hipMalloc(&d_INPUT, sizeof(INPUT));
	hipMalloc(&d_FIRST_LAYER_WEIGHT, sizeof(FIRST_LAYER_WEIGHT));
	hipMalloc(&d_FIRST_LAYER, sizeof(FIRST_LAYER));

	hipMemcpy(d_INPUT, INPUT, sizeof(INPUT), hipMemcpyHostToDevice);
	hipMemcpy(d_FIRST_LAYER_WEIGHT, FIRST_LAYER_WEIGHT, sizeof(FIRST_LAYER_WEIGHT), hipMemcpyHostToDevice);

	int numBlocks = 6;
	dim3 threadsPerBlock(13, 13);
	d_compute_first<<<numBlocks, threadsPerBlock>>>((double (*)[29])d_INPUT, (double (*)[13][13][5][5])d_FIRST_LAYER_WEIGHT, (double (*)[13][13])d_FIRST_LAYER);
	hipDeviceSynchronize();

	hipMemcpy(FIRST_LAYER, d_FIRST_LAYER, sizeof(FIRST_LAYER), hipMemcpyDeviceToHost);

	hipFree(d_INPUT);
	hipFree(d_FIRST_LAYER_WEIGHT);
	hipFree(d_FIRST_LAYER);

	// gettimeofday(&stop, NULL);
	// timersub(&stop, &start, &diff);
	// fprintf(stderr, "First layer: %ld.%06lds\n", (long int)diff.tv_sec, (long int)diff.tv_usec);
}

__host__ void compute_second() {
	// struct timeval start, stop, diff;
	// gettimeofday(&start, NULL);

	double* d_FIRST_LAYER;
	double* d_SECOND_LAYER_WEIGHT;
	double* d_SECOND_LAYER;
	
	
	hipMalloc(&d_FIRST_LAYER, sizeof(FIRST_LAYER));
	hipMalloc(&d_SECOND_LAYER_WEIGHT, sizeof(SECOND_LAYER_WEIGHT));
	hipMalloc(&d_SECOND_LAYER, sizeof(SECOND_LAYER));

	hipMemcpy(d_FIRST_LAYER, FIRST_LAYER, sizeof(FIRST_LAYER), hipMemcpyHostToDevice);
	hipMemcpy(d_SECOND_LAYER_WEIGHT, SECOND_LAYER_WEIGHT, sizeof(SECOND_LAYER_WEIGHT), hipMemcpyHostToDevice);


	int numBlocks = 50;
	dim3 threadsPerBlock(5, 5);
	d_compute_second<<<numBlocks, threadsPerBlock>>>((double (*)[13][13])d_FIRST_LAYER, (double (*)[5][5][6][5][5])d_SECOND_LAYER_WEIGHT, (double (*)[5][5])d_SECOND_LAYER);
	hipDeviceSynchronize();

	hipMemcpy(SECOND_LAYER, d_SECOND_LAYER, sizeof(SECOND_LAYER), hipMemcpyDeviceToHost);

	hipFree(d_FIRST_LAYER);
	hipFree(d_SECOND_LAYER_WEIGHT);
	hipFree(d_SECOND_LAYER);

	// gettimeofday(&stop, NULL);
	// timersub(&stop, &start, &diff);
	// fprintf(stderr, "Second layer: %ld.%06lds\n", (long int)diff.tv_sec, (long int)diff.tv_usec);
}

__host__ void compute_third() {
	// struct timeval start, stop, diff;
	// gettimeofday(&start, NULL);

	double* d_SECOND_LAYER;
	double* d_THIRD_LAYER_WEIGHT;
	double* d_THIRD_LAYER;
	
	hipMalloc(&d_SECOND_LAYER, sizeof(SECOND_LAYER));
	hipMalloc(&d_THIRD_LAYER_WEIGHT, sizeof(THIRD_LAYER_WEIGHT));
	hipMalloc(&d_THIRD_LAYER, sizeof(THIRD_LAYER));

	hipMemcpy(d_SECOND_LAYER, SECOND_LAYER, sizeof(SECOND_LAYER), hipMemcpyHostToDevice);
	hipMemcpy(d_THIRD_LAYER_WEIGHT, THIRD_LAYER_WEIGHT, sizeof(THIRD_LAYER_WEIGHT), hipMemcpyHostToDevice);

	int numBlocks = 100;
	int threadsPerBlock = 50;
	d_compute_third<<<numBlocks, threadsPerBlock>>>((double (*)[5][5])d_SECOND_LAYER, (double (*)[50][5][5])d_THIRD_LAYER_WEIGHT, (double *)d_THIRD_LAYER);
	hipDeviceSynchronize();

	hipMemcpy(THIRD_LAYER, d_THIRD_LAYER, sizeof(THIRD_LAYER), hipMemcpyDeviceToHost);

	hipFree(d_SECOND_LAYER);
	hipFree(d_THIRD_LAYER_WEIGHT);
	hipFree(d_THIRD_LAYER);

	// gettimeofday(&stop, NULL);
	// timersub(&stop, &start, &diff);
	// fprintf(stderr, "Third layer: %ld.%06lds\n", (long int)diff.tv_sec, (long int)diff.tv_usec);
}

__host__ void compute_output() {
	// struct timeval start, stop, diff;
	// gettimeofday(&start, NULL);

	double* d_THIRD_LAYER;
	double* d_OUTPUT_WEIGHT;
	double* d_OUTPUT;
	
	hipMalloc(&d_THIRD_LAYER, sizeof(THIRD_LAYER));
	hipMalloc(&d_OUTPUT_WEIGHT, sizeof(OUTPUT_WEIGHT));
	hipMalloc(&d_OUTPUT, sizeof(OUTPUT));

	hipMemcpy(d_THIRD_LAYER, THIRD_LAYER, sizeof(THIRD_LAYER), hipMemcpyHostToDevice);
	hipMemcpy(d_OUTPUT_WEIGHT, OUTPUT_WEIGHT, sizeof(OUTPUT_WEIGHT), hipMemcpyHostToDevice);

	int numBlocks = 1;
	int threadsPerBlock = 10;
	d_compute_output<<<numBlocks, threadsPerBlock>>>((double *)d_THIRD_LAYER, (double (*)[100])d_OUTPUT_WEIGHT, (double *)d_OUTPUT);
	hipDeviceSynchronize();

	hipMemcpy(OUTPUT, d_OUTPUT, sizeof(OUTPUT), hipMemcpyDeviceToHost);

	hipFree(d_THIRD_LAYER);
	hipFree(d_OUTPUT_WEIGHT);
	hipFree(d_OUTPUT);

	// gettimeofday(&stop, NULL);
	// timersub(&stop, &start, &diff);
	// fprintf(stderr, "Output layer: %ld.%06lds\n", (long int)diff.tv_sec, (long int)diff.tv_usec);
}